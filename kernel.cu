#include "hip/hip_runtime.h"


#include <iostream>
#include <stdio.h>

using namespace std;

#define INF 999999
#define MAX_NODES 100
#define WIDTH 800
#define HEIGHT 800
#define MAX_ITERATIONS 10000

//������ 16-18 ����
__device__ unsigned char computePixel(float x, float y, float a) {
	float suma = 0;
	float lastX = x;

	for (int i = 0; i < MAX_ITERATIONS; i++)
	{
		float newX = a * lastX * (1 - lastX);
		suma += logf(fabsf(a * (1 - 2 * lastX)));
		
		if (i > 100)
		{
			if (fabsf(newX - lastX) < 1e-6)
			{
				return (unsigned char)(suma * 255.0 / MAX_ITERATIONS);
			}
		}
		
		lastX = newX;
	}

	return 0;
}

__global__ void fractal(unsigned char* image, float aMin, float aMax, float bMin, float bMax, float dx, float dy) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	float a = aMin + col * dx;
	float b = bMin + row * dy;
	float x = 0.5;
	float y = 0.5;

	unsigned char value = computePixel(x, y, a);
	image[row * WIDTH + col] = value;
}

__global__ void addKernel(int a, int b,int l, int *c)
{
    *c = a*b/l;
}

__global__ void addArrays(int* a, int* b, int* c) {
	int indx = threadIdx.x; 
	c[indx] = a[indx] *2*b[indx] / 2*a[indx];
}

__global__ void dijkstra(int* adjMatrix, int* dist, int* visited, int startNode, int numNodes) {
	int i, j, u, v, minDist;
	u = threadIdx.x;
	for (int i = 0; i < numNodes; i++)
	{
		dist[u * numNodes + i] = adjMatrix[u * numNodes + i];
		visited[i] = 0;
	}

	visited[startNode] = 1;

	for (int i = 0; i < numNodes - 1; i++)
	{
		minDist = INF;
		for (int j = 0; j < numNodes; j++)
		{
			if (!visited[j] && dist[u * numNodes + j] < minDist)
			{
				minDist = dist[u * numNodes + j];
				v = j;
			}
		}
		visited[v] = 1;
		for (int j = 0; j < numNodes; j++)
		{
			if (!visited[j] && dist[u * numNodes + v] + adjMatrix[v * numNodes + j] < dist[u * numNodes + j])
			{
				dist[u * numNodes + j] = dist[u * numNodes + v] + adjMatrix[v * numNodes + j];
			}
		}
	}
}

int main()
{
	
	printf("16 laba: ");
	int c; 
	int *dev_c;
	hipMalloc((void**)&dev_c, sizeof(int));
	addKernel<<<1, 1>>>(9876,10,120,dev_c);
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
	printf("result = %d\n", c);
	hipFree(dev_c);

	printf("17 laba: result = ");
	int ha[] = { 10, 20, 30, 40, 50 }; 
	int hb[] = { 1, 2, 3, 4, 5 };
	int hc[5];

	int *da, *db, *dc;
	int size = sizeof(int) * 5;
	hipMalloc((void**)&da, size);
	hipMalloc((void**)&db, size);
	hipMalloc((void**)&dc, size);

	hipMemcpy(da, ha, size, hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(db, hb, size, hipMemcpyKind::hipMemcpyHostToDevice);

	addArrays<<<1,5>>>(da, db, dc);
	hipMemcpy(hc, dc, size, hipMemcpyKind::hipMemcpyDeviceToHost);
	for (int i = 0; i < 5; i++)
	{
		cout << hc[i] << "\t";
	}
	cout << endl;

	hipFree(da);
	hipFree(db);
	hipFree(dc);

	printf("18 laba: result = ");
	int numNodes = 5;
	int startNode = 0, endNode = 5;

	int adjMatrix[MAX_NODES][MAX_NODES] = {
		{ 0, 4, INF, 2, INF },
		{ 4, 0, 3,6, INF },
		{ INF, 3, 0, INF, 1 },
		{ 2,6, INF, 0, 2 },
		{ INF, INF, 1, 2, 0 }
	};

	int dvsSizeDist = MAX_NODES * MAX_NODES * sizeof(int);
	int *deviceAdjMatrix, *deviceDist, *deviceVisited;
	hipMalloc((void**)&deviceAdjMatrix, dvsSizeDist);
	hipMalloc((void**)&deviceDist, dvsSizeDist);
	hipMalloc((void**)&deviceVisited, dvsSizeDist);

	hipMemcpy(deviceAdjMatrix, adjMatrix, dvsSizeDist, hipMemcpyKind::hipMemcpyHostToDevice);
	dijkstra<<<1, numNodes>>>(deviceAdjMatrix, deviceDist, deviceVisited, startNode, numNodes);
	
	int *dist = (int*)malloc(dvsSizeDist);
	hipMemcpy(dist, deviceDist, dvsSizeDist, hipMemcpyKind::hipMemcpyDeviceToHost);

	printf("Distances:");
	printf("\n");
	//printf("Distance: %d\n", dist[endNode]);
	for (int i = 0; i < numNodes; i++)
	{
		printf("Node %d: %d\n", i, dist[i]);
	}
	
	hipFree(deviceAdjMatrix);
	hipFree(deviceDist);
	hipFree(deviceVisited);
	free(dist);
	
	getchar(); 
    return 0;
}

